#include "hip/hip_runtime.h"

// compile with "nvcc SSFM.cu functions.cu -lcufftw -lcufft -lcurand -o SSFMcu"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include <hiprand.h>
#include "common.h"

// Complex data type
typedef hipfftDoubleComplex CC;

#define PI2 2.0 * 3.14159265358979323846  //2*pi
#define C 299792458*1E9/1E12  // speed of ligth in vacuum [nm/ps]
#define NELEMS(x)  (sizeof(x) / sizeof((x).x)) // number of elements of an array

#include "functions.h"

int main(int argc, char *argv[]){
	int i; 
	int N = 1<<16; // number of points
	
	// parameters for kernels
	int dimx = 1 << 5;
	dim3 block(dimx);
	dim3 grid((N + block.x - 1) / block.x);
	
	// set up device	
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));
	
	int nBytes =  sizeof(CC)*N;
	double t_width = (double )(N * 0.003); // time window size
	double T0 = 0.1; //temporal width of pulses [ps]
	double lambda_0 = 5000; // central wavelength [nm]
        double w0 = PI2 * C / lambda_0; // angular frequency in 2*pi*[THz]
	double betas[3] = {-0.05, 0.004,0}; // betas [ps^i / km]
	int lb = 3; // number of betas that are included
	double dT, dF; // step time and frequency
	double sol_ord, P0, factor; // soliton order, power and factor for anomalous dispersion
	
	double gamma = 0.1; // nonlinear parameter gamma [1/W/m]
	float tau1 = 0.0155, tau2 = 0.2305; // Raman times [ps] 
	float fr = 0.1; // fractional Raman contribution
	
        char kindpower = 'p'; // select among different kind of power
	switch(kindpower) {
        case 'n': // select soliton order and then associated power will be computed
            sol_ord = 1; // Soliton order
            P0 = pow(sol_ord,2) * fabs(betas[0])/(gamma*pow(T0,2));
            break;
        case 'p': // select power and then soliton order will be computed
            factor = 0.7; // normalized power from cutoff
            P0 = (fabs(betas[0])* w0 * w0/gamma)*factor; // peak power of input [W]
            sol_ord = sqrt(P0*gamma*pow(T0,2)) / fabs(betas[0]);
            break;
        case 'a': // arbitrary power
            double P0 = 50.00; // peak power of input [W]
            break;
    }
	
		
    /* Distances */
	double LD = pow(T0,2) / fabs(betas[0]);  // dispersion lenght
	// double LD3 = pow(T0,3) / fabs(betas[2]); // third order dispersion length
    double LNL = 1/gamma/P0; // nonlinear length
    double Zfiss = LD/sol_ord; // soliton fission length
    double Zsol = 0.5 * 3.14159265358979323846 * LD; // soliton period
	double flength = 0.0012;
	double h = flength/4000; // z step
	int steps_z = (int )floor(flength/h); // number of steps in Z

	/* Set plan for cuFFT */
	hipfftHandle plan_1;
	hipfftPlan1d(&plan_1, N, HIPFFT_Z2Z, 1);
	
	CC *u1 = (CC*)malloc(nBytes);	CC *u1_W = (CC*)malloc(nBytes);
	CC *u2 = (CC*)malloc(nBytes);	CC *u2_W = (CC*)malloc(nBytes);
	CC *u3 = (CC*)malloc(nBytes);	CC *u3_W = (CC*)malloc(nBytes);
	CC *u4 = (CC*)malloc(nBytes);	CC *u4_W = (CC*)malloc(nBytes);
	CC *u_ip = (CC*)malloc(nBytes);	CC *D_OP = (CC*)malloc(nBytes);     // Linear operator exp(Dh/2)
	CC *hR = (CC*)malloc(nBytes);  // Raman response in time domain
	CC *hR_W = (CC*)malloc(nBytes);     // Raman response in frequency domain
	CC *self_st = (CC*)malloc(nBytes);  // Self-steepening
	CC *alpha1 = (CC*)malloc(nBytes);  CC *alpha2 = (CC*)malloc(nBytes);
	CC *alpha3 = (CC*)malloc(nBytes);  CC *alpha4 = (CC*)malloc(nBytes);
	CC *V_ss = (CC*)malloc(nBytes);
	/*************************/    

	/* Time, frequency and Z vectors*/
	double *T;    
	T = (double*) malloc(sizeof(double) * N);
	dT = inic_vector_T(T, N, t_width);

	double *TT;    
	TT = (double*) malloc(sizeof(double) * N);
	inic_vector_Traman(TT, N, t_width);
	
	double *V;    
	V = (double*) malloc(sizeof(double) * N);
	dF = inic_vector_F(V, N, dT);
	
	freq_shift( V_ss, V, N ); //frequecy used in DOP and self-steepening

	double *Z;
	Z = (double*) malloc(sizeof(double) * steps_z);
	inic_vector_Z(Z, steps_z, h);
	
	/*************************/
	/* Set Raman */
	
	/* Raman */
	RAMAN_RESP(hR, N, tau1, tau2, TT);
	CC *d_hR, *d_hR_W;
	CHECK(hipMalloc((void **)&d_hR, nBytes));
	CHECK(hipMalloc((void **)&d_hR_W, nBytes));
	CHECK(hipMemset(d_hR, 0, nBytes));
	CHECK(hipMemset(d_hR_W, 0, nBytes));
	// Copy host memory to device
	CHECK(hipMemcpy(d_hR, hR, nBytes, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(d_hR_W, hR_W, nBytes, hipMemcpyHostToDevice));
	hipfftExecZ2Z(plan_1, (CC *)d_hR, (CC *)d_hR_W, HIPFFT_BACKWARD);
	hipDeviceSynchronize();
	//Scale Raman W
	scale<<<grid,block>>>(d_hR_W, N, dT);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());    
	CHECK(hipMemcpy(hR_W, d_hR_W, nBytes, hipMemcpyDeviceToHost));
	
	/*************************/
	
	/* Self steepening */
	inic_selfst(self_st, V_ss, w0, gamma, N);
	/*******************/

	printf("N = %i points\nbeta2 = %f ps^2/km\ndT = %f ps\ndF = %f THz\nfr = %f\ntau1 = %f ps\ntau2 = %f ps\nw0 = %.2f THz\nlambda0 = %.1f nm\nstep size = %f m\nDistance = %f m\nPower = %.2f W\np = %0.2f\n",N, betas[0], dT, dF, fr, tau1, tau2, w0, lambda_0, h,flength, P0, factor);
	
	/* Input field and envelope expressed in the interaction picture */    
	linear_operator(D_OP, V_ss, betas, lb, N, h); //set exp(D*h/2) as a function of omega = 2*pi*f
	unsigned int NOISE = 1;
	if (NOISE == 0){
		input_field_T(u1, T, N, T0, P0); // signal without noise
	}
	else{
		input_field_T(u1, T, N, T0, P0);
		CC *h_noise = (CC *)malloc(nBytes);
		double SNR = 30; // Signal-to-Noise ratio
		noise_generator(h_noise, SNR, N, P0 );
		for (int j = 0; j < N; j++){
			u1[j].x = u1[j].x + h_noise[j].x;
			u1[j].y = u1[j].y + h_noise[j].y;
		}
		free(h_noise);
	}
	
	/* Device vectors */
	CC *d_u_ip, *d_alpha1, *d_alpha2, *d_alpha3, *d_alpha4, *d_u1_W, *d_u1, *d_u2_W, *d_u2, *d_u3_W, *d_u3, *d_u4_W, *d_u4;
	CHECK(hipMalloc((void **)&d_u1_W, nBytes)); CHECK(hipMemset(d_u1_W, 0, nBytes));
	CHECK(hipMalloc((void **)&d_u1, nBytes)); CHECK(hipMemset(d_u1, 0, nBytes));
	CHECK(hipMemcpy(d_u1, u1, nBytes, hipMemcpyHostToDevice));    
    
	/* computes FFT for input field */
	hipfftExecZ2Z(plan_1, (CC *)d_u1, (CC *)d_u1_W, HIPFFT_BACKWARD);
	CHECK(hipDeviceSynchronize());
	CUFFTscale<<<grid,block>>>(d_u1_W, N, N);
	CHECK(hipDeviceSynchronize());
	CHECK(hipGetLastError());
	CHECK(hipMemcpy(u1_W, d_u1_W, nBytes, hipMemcpyDeviceToHost));
	
	
	/* Saving some vectors */
	FILE *uno;	
	uno = fopen("T.txt", "w+");
	for ( int i = 0; i < N; i++ )
		fprintf(uno, "%15.10f\t", T[i]);// writing data into file
	fclose(uno);//closing file

	FILE *dos;
	dos = fopen("V.txt", "w+");
	for ( int i = 0; i < N; i++ )
		fprintf(dos, "%15.10f\t", V[i]);// writing data into file
	fclose(dos);//closing file
	
	
	/* Allocating memory on GPU */
	CHECK(hipMalloc((void **)&d_u_ip, nBytes));
	CHECK(hipMalloc((void **)&d_alpha1, nBytes));
	CHECK(hipMalloc((void **)&d_alpha2, nBytes));
	CHECK(hipMalloc((void **)&d_alpha3, nBytes));
	CHECK(hipMalloc((void **)&d_alpha4, nBytes));
	CHECK(hipMalloc((void **)&d_u2_W, nBytes));
	CHECK(hipMalloc((void **)&d_u2, nBytes)); 
	CHECK(hipMalloc((void **)&d_u3_W, nBytes));
	CHECK(hipMalloc((void **)&d_u3, nBytes)); 
	CHECK(hipMalloc((void **)&d_u4_W, nBytes));
	CHECK(hipMalloc((void **)&d_u4, nBytes)); 
	/*************************/	
	
	printf("Starting main loop on CPU & GPU...\n");
	double iStart = seconds();
	
    /* START MAIN LOOP */   
	for (int s = 1; s < steps_z; s++){
		cpx_prod (D_OP, u1_W, u_ip, N); // A_I(w,z) = exp(D*h/2)*A(w,z)
		
		COMPUTE_TFN( alpha1, u1, u1_W, hR_W, V, self_st, N, fr, nBytes );
		
		CC *aux1 =  (CC*)malloc(nBytes);
		cpx_prod (D_OP, alpha1, aux1, N);
		for (i = 0; i<N; i++){
			alpha1[i].x = aux1[i].x;
			alpha1[i].y = aux1[i].y;        
		}
		free(aux1);
			
		CHECK(hipMemcpy(d_u_ip, u_ip, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_alpha1, alpha1, nBytes, hipMemcpyHostToDevice));
		lineal<<<grid,block>>>(d_u_ip, d_alpha1, d_u2_W, N, h/2);

		CHECK(hipMemcpy(u2_W, d_u2_W, nBytes, hipMemcpyDeviceToHost));
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());
		hipfftExecZ2Z(plan_1, (CC *)d_u2_W, (CC *)d_u2, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		CHECK(hipMemcpy(u2, d_u2, nBytes, hipMemcpyDeviceToHost));
		COMPUTE_TFN( alpha2, u2, u2_W, hR_W, V, self_st, N, fr, nBytes );
		CHECK(hipMemcpy(d_alpha2, alpha2, nBytes, hipMemcpyHostToDevice));
		lineal<<<grid,block>>>(d_u_ip, d_alpha2, d_u3_W, N, h/2);  

		CHECK(hipMemcpy(u3_W, d_u3_W, nBytes, hipMemcpyDeviceToHost));
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());     

		hipfftExecZ2Z(plan_1, (CC *)d_u3_W, (CC *)d_u3, HIPFFT_FORWARD);
		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError()); 
		CHECK(hipMemcpy(u3, d_u3, nBytes, hipMemcpyDeviceToHost));

		COMPUTE_TFN( alpha3, u3, u3_W, hR_W, V, self_st, N, fr, nBytes );
		CHECK(hipMemcpy(d_alpha3, alpha3, nBytes, hipMemcpyHostToDevice));        
		CC *aux2 =  (CC*)malloc(nBytes);
		CC *d_aux2; CHECK(hipMalloc((void **)&d_aux2, nBytes));  CHECK(hipMemset(d_aux2, 0, nBytes));
		lineal<<<grid,block>>>(d_u_ip, d_alpha3, d_aux2, N, h); 

		CHECK(hipDeviceSynchronize());
		CHECK(hipGetLastError());    
		CHECK(hipMemcpy(aux2, d_aux2, nBytes, hipMemcpyDeviceToHost));       
		cpx_prod (D_OP, aux2, u4_W, N);
		free(aux2); CHECK(hipFree(d_aux2));
		
		CHECK(hipMemcpy(d_u4_W, u4_W, nBytes, hipMemcpyHostToDevice));
		hipfftExecZ2Z(plan_1, (CC *)d_u4_W, (CC *)d_u4, HIPFFT_FORWARD);
		hipDeviceSynchronize();
		CHECK(hipMemcpy(u4, d_u4, nBytes, hipMemcpyDeviceToHost));

		COMPUTE_TFN( alpha4, u4, u4_W, hR_W, V, self_st, N, fr, nBytes );

		CC *aux3 =  (CC*)malloc(nBytes);
		CC *aux4 =  (CC*)malloc(nBytes);
		
		CC *d_aux3; CHECK(hipMalloc((void **)&d_aux3, nBytes)); CHECK(hipMemset(d_aux3, 0, nBytes));
		CC *d_aux4; CHECK(hipMalloc((void **)&d_aux4, nBytes)); CHECK(hipMemset(d_aux4, 0, nBytes));

		final<<<grid,block>>>(d_u_ip, d_alpha1, d_alpha2, d_alpha3, d_aux3, h, N); 

		CHECK(hipMemcpy(aux3, d_aux3, nBytes, hipMemcpyDeviceToHost));
		cpx_prod (D_OP, aux3, aux4, N);

		free(aux3); CHECK(hipFree(d_aux3));
		
		CHECK(hipMemcpy(d_aux4, aux4, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemcpy(d_alpha4, alpha4, nBytes, hipMemcpyHostToDevice));
		CHECK(hipMemset(d_u1_W, 0, nBytes));
		lineal<<<grid,block>>>(d_aux4, d_alpha4, d_u1_W, N, h/6);

		free(aux4); CHECK(hipFree(d_aux4));
		
		CHECK(hipMemcpy(u1_W, d_u1_W, nBytes, hipMemcpyDeviceToHost));

		hipfftExecZ2Z(plan_1, (CC *)d_u1_W, (CC *)d_u1, HIPFFT_FORWARD);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(u1, d_u1, nBytes, hipMemcpyDeviceToHost));
		
		//printf("%.2f %% completed...\n", (double) s*100/(steps_z-1));   
		
		CHECK(hipMemset(d_alpha1, 0, nBytes));
		CHECK(hipMemset(d_alpha2, 0, nBytes));
		CHECK(hipMemset(d_alpha3, 0, nBytes));
		CHECK(hipMemset(d_alpha4, 0, nBytes));
		CHECK(hipMemset(d_u1_W, 0, nBytes));	
		CHECK(hipMemset(d_u1, 0, nBytes));/////////
		CHECK(hipMemset(d_u2_W, 0, nBytes));
		CHECK(hipMemset(d_u2, 0, nBytes));/////////
		CHECK(hipMemset(d_u3_W, 0, nBytes));
		CHECK(hipMemset(d_u3, 0, nBytes));/////////
		CHECK(hipMemset(d_u4_W, 0, nBytes));
		CHECK(hipMemset(d_u4, 0, nBytes));/////////
		}
		FILE *tres, *cuatro;
		tres = fopen("output_T.txt", "w+");
		cuatro = fopen("output_W.txt", "w+");
		for ( i = 0; i < N; i++ ){
			fprintf(tres, "%15.10f\t%15.10f\n", u1[i].x, u1[i].y);// writing data into file
		fprintf(cuatro, "%15.10f\t%15.10f\n", u1_W[i].x, u1_W[i].y);// writing data into file
	}
	fclose(tres);//closing file	
	fclose(cuatro);//closing file
	

	double iElaps = seconds() - iStart;
	printf("...time elapsed %f sec\n", iElaps);
	
	/* Deallocating memory and destroying plans */
	free(u1); free(u1_W); free(u2); free(u2_W);
	free(u3); free(u3_W); free(u4); free(u4_W);
	free(alpha1); free(alpha2); free(alpha3); free(alpha4);
	free(u_ip); free(D_OP); free(self_st); free(hR);
	free(hR_W); free(V_ss); free(T); free(TT);
	free(V); free(Z);
	CHECK(hipFree(d_u_ip)); CHECK(hipFree(d_alpha1));
	CHECK(hipFree(d_alpha2)); CHECK(hipFree(d_alpha3));
	CHECK(hipFree(d_u1)); CHECK(hipFree(d_u2));
	CHECK(hipFree(d_u3)); CHECK(hipFree(d_u4));
	CHECK(hipFree(d_u1_W)); CHECK(hipFree(d_u2_W));
	CHECK(hipFree(d_u3_W)); CHECK(hipFree(d_u4_W));
	//Destroy CUFFT context
	hipfftDestroy(plan_1);
	
	hipDeviceReset();

	return 0;
}