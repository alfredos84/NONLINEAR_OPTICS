#include "hip/hip_runtime.h"
// Compilation line:        nvcc cwOPO.cu --gpu-architecture=sm_75 -lcufftw -lcufft -o cuOPO
//
// Consider changing the flag sm_75 by the proper one based in the GPU architecture



/**
 * This main file computes the evolution of the electric field
 * at the signal and pump frequency for a degenerate optical 
 * parametric oscillator. The code includes chirped mirrors
 * comensating for the group-velocity dispersion as well as
 * an intracavity electro-optical modulator.
 * 
 * The code receives a set of parameteres passed as an 
 * external arguments. For this propose, it is convenient to use
 * a shell script file in order to nicely manupulate the data.
 */

#include <iostream>
#include <string>
#include <random>
#include <fstream>
#include <iomanip>
#include <typeinfo>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <stdio.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/host_vector.h>
#include <thrust/complex.h>
#include <hip/hip_complex.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/generate.h>
#include <thrust/transform.h>
#include <thrust/random.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>

#include "SaveFiles.cuh"
#include "common.cuh"
#include "fft.cuh"
#include "refindex.cuh"
#include "functors.cuh"
#include "operators.cuh"
#include "functions.cuh"




/** 
 * Complex data type: a set of datatypes are
 * defined to make the code more readable.
 *
 * Definitions for numbers
 * typefl_t : datatype for real numbers
 * CC_t     : datatype for complex numbers
 * 
 * Definitions for vectors:
 * 
 * rVech_t  : real vector host
 * rVecd_t  : real vector device
 * cVech_t  : complex vector host
 * cVecd_t  : complex vector device
 */

using typefl_t = float;
using CC_t = hipFloatComplex;
using rVech_t = thrust::host_vector<typefl_t>;
using rVecd_t = thrust::device_vector<typefl_t>;
using cVech_t = thrust::host_vector<CC_t>;
using cVecd_t = thrust::device_vector<CC_t>;	
	

int main(int argc, char *argv[]){
	
	
	const typefl_t PI   = 3.141592653589793238462643383279502884;	
	const typefl_t C    = 299792458*1E6/1E12;              // speed of ligth in vacuum [um/ps]
	const typefl_t EPS0 = 8.8541878128E-12*1E12/1E6;       // vacuum pertivity [W.ps/V²μm] 

	std::cout << "\n\n\n#######---Welcome to OPO calculator---#######\n\n\n" << std::endl;

	time_t current_time;
	time(&current_time);
	std::cout << ctime(&current_time) << std::endl;
	
	// timing the code
	double iStart = seconds();

	std::string Filename, SAux, Extension = ".dat";
	
	
	// Set up device //
	int dev = 0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	std::cout << "\n\nUsing Device " << dev << ": GPU " << deviceProp.name << std::endl;
	CHECK(hipSetDevice(dev));

	std::cout << "Setting constants and vectors in host..." << std::endl;
	
	// Set parameters and constants
	int N_rt                 = atoi(argv[8]); // number of round trips to cover the input pulse

	// Define wavelengths
	typefl_t lp              = 0.532;    // pump wavelength [μm]
	typefl_t ls              = 2*lp;     // signal wavelength [μm]

	typefl_t Temperature     = 27;       // Crystal temperature [ºC]
	typefl_t deff            = 14.77e-6; // effective d [um/V]
	typefl_t Lambda          = 6.97;     // grating period [um]
	typefl_t alphap          = 0.002e-4; // pump linear absorption [1/um]
	typefl_t alphas          = 0.025e-4; // signal linear absorption [1/um]
	typefl_t np              = n_PPLN(lp, Temperature); // refractive index at pump wavelength
	typefl_t vp              = group_vel_PPLN(lp, Temperature); // group velocity at pump wavelength
	typefl_t b2p             = gvd_PPLN(lp, Temperature);  // GVD at pump wavelength
	typefl_t kp              = 2*PI*deff/(n_PPLN(lp, Temperature)*lp); // kappa pump [1/V]
	
	typefl_t ns              = n_PPLN(ls, Temperature);  // refractive index at signal wavelength
	typefl_t vs              = group_vel_PPLN(ls, Temperature); // group velocity at signal wavelength
	typefl_t b2s             = gvd_PPLN(ls, Temperature); // GVD at signal wavelength
	typefl_t ks              = 2*PI*deff/(n_PPLN(ls, Temperature)*ls); // kappa signal [1/V]
	typefl_t dk              = 2*PI*( np/lp - 2*ns/ls - 1/Lambda ); // mismatch factor
	
	typefl_t Lcr             = 5e3;  // crystal length [um]
	typefl_t Lcav            = atof(argv[4]) * Lcr;  // cavity length [um]
	typefl_t R               = atof(argv[5])*0.01;  // net reflectivity 
	typefl_t t_rt            = (Lcav+Lcr*(n_PPLN(ls, Temperature)-1))/C;
	typefl_t FSR             = 1/t_rt; // free spectral range
	typefl_t delta           = atof(argv[6]); if(R<=0.5){delta *= 0.01;} else{delta *= 0.001;}
	typefl_t epsilon         = atof(argv[7])*0.01;
	typefl_t GDD             = -epsilon*b2s*Lcr; // GDD [ps²]

	int steps_z              = atoi(argv[3]); // number of z step inside the crystal
	typefl_t dz              = Lcr/steps_z;   // z-step size
	
	// Time and frequency discretization
	
	unsigned int ex          = atoi(argv[2]);
	int N_ps                 = 1 << ex;  // points per time slice
	typefl_t dT              = t_rt/N_ps; // time step in [ps]
	typefl_t dF              = 1/t_rt; // frequency step in [THz]
	int SIZE                 = N_ps;
	unsigned int Nrts        = 256; // number of round trips to save
	int SIZEL                = N_ps*Nrts;
	typefl_t T_width         = (typefl_t ) (Nrts*t_rt); // total time for input ns-pulse
    
	/* vector T for one round trip */
	rVech_t T(SIZE); 
	linspace( T, -0.5*t_rt, 0.5*t_rt);

	/* vectors Tp for the complete time */
	rVech_t  Tp(SIZEL); 
	linspace( Tp, -0.5*T_width, 0.5*T_width);
	
	/* vector F_p for the complete pump frequency */
	typefl_t dF_p = 1/T_width;
	rVech_t F_p(SIZEL) ; inic_vector_F(F_p, dF_p);

	short unsigned int save_vectors = atoi(argv[1]);
	if (save_vectors == 1){
		std::cout << "\nSaving time and frequency vectors...\n" << std::endl;
		Filename = "Tp"; SaveFileVectorReal (Tp, Filename+Extension);
 		Filename = "freq"; SaveFileVectorReal (F_p, Filename+Extension);
		Filename = "T"; SaveFileVectorReal (T, Filename+Extension);
	}
	else{ std::cout << "\nTime and frequency were previuosly save...\n" << std::endl;}
	
	rVech_t F_ext(SIZE); inic_vector_F(F_ext, dF); // extended freq. grid [THz]

	rVech_t w(SIZE);  // angular frequency Ω
	cVech_t w_GVDp_h(SIZE); // e^(i.dz.((1/vs-1/vp).Ω + ½.β.Lcr.Ω²))
	cVech_t w_GVDs_h(SIZE); // i.½.β.Ω².dz
	cVech_t w_Comp_h(SIZE);  // e^(-i.½.ε.β.Lcr.Ω²)
	
	fftshift(w, F_ext);   // define ang ref for FFTs  [2*pi*THz]
	
	for ( int i = 0; i < w.size(); i++ ){
		w[i] *= 2*PI;
		w_GVDp_h[i].x = +cosf( dz * (w[i]*(1/vs-1/vp) + 0.5*b2p*w[i]*w[i]) );
		w_GVDp_h[i].y = +sinf( dz * (w[i]*(1/vs-1/vp) + 0.5*b2p*w[i]*w[i]) );
		w_GVDs_h[i].x = +cosf( dz * 0.5*b2s*w[i]*w[i] );
		w_GVDs_h[i].y = +sinf( dz * 0.5*b2s*w[i]*w[i] );
		w_Comp_h[i].x = +cosf( 0.5*GDD*w[i]*w[i] );
		w_Comp_h[i].y = +sinf( 0.5*GDD*w[i]*w[i] );
	}
	
	cVecd_t w_GVDp_d = w_GVDp_h, w_GVDs_d = w_GVDs_h, w_Comp_d = w_Comp_h; // copy to device the angular frequency vectors
	
		
	// Pumping parameters //
	
	std::string pump_regime = "cw";					// continuous wave pump
	typefl_t waist           = 55;                      		// beam waist radius [um]
	typefl_t spot            = PI*waist*waist;          		// spot area [μm²]
	typefl_t Power           = atof(argv[9])*1e-3;           	// pump power [mW]
	typefl_t Ap0             = sqrt(2*Power/(spot*np*EPS0*C)) ; // input field amplitud [V/μm]
	
	cVech_t Ap_in(SIZE);	
	InputField(Ap_in, Ap0, pump_regime);
	
	// Signal vector (is a complex noisy vector)
	cVech_t As(SIZE);	NoiseGeneratorCPU ( As );
	
	Filename = "signal_input";
	SaveFileVectorComplex(As, Filename);

	bool prt_param_onscreen = true;
	if( prt_param_onscreen ){
		// Print parameters
		std::cout << "\n\nSimulation parameters:\n\n " << std::endl;
		std::cout << "Number of round trips   = " << N_rt  << std::endl;
		std::cout << "Pump wavelength         = " << lp*1e3 << " nm" << std::endl;
		std::cout << "Signal wavelength       = " << ls*1e3 << " nm" << std::endl;
		std::cout << "Temperature             = " << Temperature << " ºC" << std::endl;
		std::cout << "np                      = " << np << std::endl;
		std::cout << "ns                      = " << ns << std::endl;
		std::cout << "\u03BD⁻¹ pump                = " << 1.0/vp << " ps/\u03BCm" << std::endl;
		std::cout << "\u03BD⁻¹ signal              = " << 1.0/vs << " ps/\u03BCm" << std::endl;
		std::cout << "\u0394k                      = " << dk << " \u03BCm⁻¹" << std::endl;
		std::cout << "GVD pump                = " << b2p << " ps²/\u03BCm" << std::endl;
		std::cout << "GVD signal              = " << b2s << " ps²/\u03BCm" << std::endl;
		std::cout << "GVD compensation        = " << atoi(argv[7]) << " %"  << std::endl;
		std::cout << "Cavity net dispersion   = " << (1-epsilon)*b2s*1e3 << " fs²/\u03BCm"  << std::endl;
		std::cout << "deff                    = " << deff*1e6 << " pm/V"  << std::endl;
		std::cout << "\u039B                       = " << Lambda << " \u03BCm"  << std::endl;
		std::cout << "\u03B1p                      = " << alphap << " \u03BCm⁻¹"  << std::endl;
		std::cout << "\u03B1s                      = " << alphas << " \u03BCm⁻¹" << std::endl;
		std::cout << "Crystal length          = " << Lcr*1e-3 << " mm"  << std::endl;
		std::cout << "Cavity  length          = " << Lcav*1e-3 << " mm"  << std::endl;
		std::cout << "\u0394z                      = " << dz << " \u03BCm"  << std::endl;
		std::cout << "Reflectivity            = " << R*100 << " %"  << std::endl;	
		std::cout << "Round-trip time         = " << std::setprecision(15) << t_rt << " ps"  << std::endl;	
		std::cout << "FSR                     = " << std::setprecision(15) << FSR*1e3 << " GHz"  << std::endl;
		std::cout << "Cavity detuning (\u03B4)     = " << delta << "\u03C0"  << std::endl;	
		std::cout << "Using N                 = 2^" << ex << " = " << N_ps << " points" << std::endl;
		std::cout << "dT                      = " << dT << " ps" << std::endl;
		std::cout << "SIZEL                   = " << SIZEL << std::endl;
		std::cout << "dF_p                    = " << dF_p << " THz" << std::endl;
		std::cout << "Ap0                     = " << Ap0 << " V/um" << std::endl; 
		std::cout << "waist                   = " << waist << " \u03BCm" << std::endl;
		std::cout << "spot                    = " << spot << " \u03BCm²" << std::endl;
		std::cout << "Power                   = " << Power << " W" << std::endl;
	}

	/********************************/
	// PHASE MODULATION
	bool using_phase_modulator = atoi(argv[10]);
	typefl_t mod_depth, fpm;
	cVech_t T_PM_h(SIZE);
	if(using_phase_modulator){
		mod_depth       = atof(argv[11])*0.1;
		fpm             = FSR - atof(argv[12])*1e-6;
		
		std::cout << "\n\nUsing a phase modulator:" << std::endl;
		std::cout << "Mod. depth (\u03B2)          = " << mod_depth << std::endl;
		std::cout << "Mod. frequency (fpm)    = " << fpm*1e3 << " GHz" << std::endl;
	}
	else{std::cout << "No phase modulator" << std::endl;}
	
	for (int i = 0; i < T_PM_h.size(); i++){
		T_PM_h[i].x = cosf(mod_depth*sinf(2*PI*fpm*T[i]));
		T_PM_h[i].y = sinf(mod_depth*sinf(2*PI*fpm*T[i]));
	}
	
	cVecd_t T_PM_d = T_PM_h;		
	/********************************/
	
	/********************************/
	// Device vectors	//    
	cVecd_t As_d = As, 	Asw_d(SIZE),	As_total_d;   // signal device vectors
	cVecd_t Ap_d(SIZE),	Apw_d(SIZE);	              // pump device vectors

	
	// Auxiliar device vectors //
	cVecd_t k1p(SIZE), k2p(SIZE), k3p(SIZE), k4p(SIZE), auxp(SIZE);
	cVecd_t k1s(SIZE), k2s(SIZE), k3s(SIZE), k4s(SIZE), auxs(SIZE);
	/********************************/
	
	
	// Set plan for cuFFT 1D and 2D//
	hipfftHandle plan;		hipfftPlan1d(&plan, SIZE, HIPFFT_C2C, 1);
	
	std::cout << "Starting main loop on GPU...\n" << std::endl;
	for (int nn = 0; nn < N_rt; nn++){
		if( nn%500 == 0 or nn == N_rt-1 )
			std::cout << "#round trip: " << nn << std::endl;

		Ap_d = Ap_in; // In every round trip, Ap <- Bin

		// Evolution along the nonlinear crystal
		SinglePass( plan, w_GVDp_d, w_GVDs_d, Ap_d,
				As_d, Apw_d, Asw_d, k1p, k1s,
				k2p, k2s, k3p, k3s, k4p, k4s,
				auxp, auxs,	dk, alphap, alphas,
				kp, ks, dz, steps_z );

		ifft ( As_d, Asw_d, plan );
// 		thrust::transform( w_Comp_d.begin(), w_Comp_d.end(), Asw_d.begin(), Asw_d.begin(), ComplexMult());
		Asw_d *= w_Comp_d;		
		fft ( Asw_d, As_d, plan );
		
		AddPhase( As_d, R, delta, nn );	// Add phase and loss (R)
		
		if( using_phase_modulator )		// Add intracavity phase modulator
			As_d *= T_PM_d;
		
		if (nn >= N_rt - Nrts)			// Accumulates several round trips to save them
			As_total_d.insert(As_total_d.end(),
						As_d.begin(), As_d.end());
	}
	
	// Destroy CUFFT context //
	hipfftDestroy(plan); 
	
	
	
	// Save outputs //
	cVech_t As_total = As_total_d;
	std::cout << "Saving outputs..." << std::endl;
	Filename = "signal_total_delta_", SAux = std::to_string(delta);
	SaveFileVectorComplex (As_total, Filename+SAux);
	
	As = As_d;
	Filename = "signal_last_rt";
	SaveFileVectorComplex (As, Filename);
	
	cVech_t Ap = Ap_d;
	Filename = "pump_last_rt";
	SaveFileVectorComplex (Ap, Filename);
	
	
	
	// Finish timing
	double iElaps = seconds() - iStart;
	if(iElaps>60){std::cout << "\n\n...time elapsed " <<  iElaps/60.0 << " min\n\n " << std::endl;}
	else{std::cout << "\n\n...time elapsed " <<  iElaps << " min\n\n " << std::endl;}

	time(&current_time);
	std::cout << ctime(&current_time) << std::endl;
	
	return 0;
	
}


/**
 Letter   Description  Escape-Sequence
-------------------------------------
A        Alpha        \u0391
B        Beta         \u0392
Γ        Gamma        \u0393
Δ        Delta        \u0394
Ε        Epsilon      \u0395
Ζ        Zeta         \u0396
Η        Eta          \u0397
Θ        Theta        \u0398
Ι        Iota         \u0399
Κ        Kappa        \u039A
Λ        Lambda       \u039B
Μ        Mu           \u039C
Ν        Nu           \u039D
Ξ        Xi           \u039E
Ο        Omicron      \u039F
Π        Pi           \u03A0
Ρ        Rho          \u03A1
Σ        Sigma        \u03A3
Τ        Tau          \u03A4
Υ        Upsilon      \u03A5
Φ        Phi          \u03A6
Χ        Chi          \u03A7
Ψ        Psi          \u03A8
Ω        Omega        \u03A9 
-------------------------------------
Letter   Description  Escape-Sequence
-------------------------------------
α        Alpha        \u03B1
β        Beta         \u03B2
γ        Gamma        \u03B3
δ        Delta        \u03B4
ε        Epsilon      \u03B5
ζ        Zeta         \u03B6
η        Eta          \u03B7
θ        Theta        \u03B8
ι        Iota         \u03B9
κ        Kappa        \u03BA
λ        Lambda       \u03BB
μ        Mu           \u03BC
ν        Nu           \u03BD
ξ        Xi           \u03BE
ο        Omicron      \u03BF
π        Pi           \u03C0
ρ        Rho          \u03C1
σ        Sigma        \u03C3
τ        Tau          \u03C4
υ        Upsilon      \u03C5
φ        Phi          \u03C6
χ        Chi          \u03C7
ψ        Psi          \u03C8
ω        Omega        \u03C9
-------------------------------------
*/
